#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define CUDA_VERSION 8000  // GLM works with ver higher than 8.0
#define GLM_FORCE_CUDA
#define GLM_FORCE_RADIANS
#define GLM_FORCE_DEPTH_ZERO_TO_ONE  /// coerce the perspective projection matrix to be in depth: [0.0 to 1.0]
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/quaternion.hpp>

#include "MD5CudaAnimation.h_cu"

namespace md5_cuda_animation {
#ifdef __HIPCC__
struct __align__(16) MD5Vertex
#else
struct alignas(16) MD5Vertex
#endif
{
    // gpu data
    uint32_t gpuVertexIndex;
    // for internal using
    int startWeight;
    int weightCount;
};

#ifdef __HIPCC__
struct __align__(16) Joint
#else
struct alignas(16) Joint
#endif
{
    int parentID;

    glm::vec3 pos;
    glm::quat orientation;
};

#ifdef __HIPCC__
struct __align__(16) BoundingBox
#else
struct alignas(16) BoundingBox
#endif
{
    glm::vec3 min;
    glm::vec3 max;
};


#ifdef __HIPCC__
    struct __align__(16) FrameData
#else
    struct alignas(16) FrameData
#endif
{
    int frameID;
    float* frameData;
    uint32_t frameDataCount;
};

#ifdef __HIPCC__
    struct __align__(16) AnimJointInfo
#else
    struct alignas(16) AnimJointInfo
#endif
{
    int parentID;

    int flags;
    int startIndex;
};

#ifdef __HIPCC__
    struct __align__(16) ModelAnimation
#else
    struct alignas(16) ModelAnimation
#endif
{
    int numFrames;
    int numJoints;
    int frameRate;
    int numAnimatedComponents;

    float frameTime;
    float totalAnimTime;
    float currAnimTime;

    AnimJointInfo* jointInfo;
    uint32_t jointInfoCount;
    BoundingBox* frameBounds;
    uint32_t frameBoundsCount;
    Joint* baseFrameJoints;
    uint32_t baseFrameJointsCount;
    FrameData* frameData;
    uint32_t frameDataCount;
    Joint** frameSkeleton;
    uint32_t frameSkeletonCount;
};

#ifdef __HIPCC__
    struct __align__(16) Weight
#else
    struct alignas(16) Weight
#endif
{
    int jointID;
    float bias;
    glm::vec3 pos;
    glm::vec3 normal;
};

#ifdef __HIPCC__
    struct __align__(16) ModelSubset
#else
    struct alignas(16) ModelSubset
#endif
{
    int numTriangles;
    uint32_t realMaterialId{0u};
    uint32_t indexOffset{0u};
    uint32_t vertOffset{0u};

    VertexData* gpuVertices;
    uint32_t gpuVerticesCount;
    MD5Vertex* vertices;
    uint32_t verticesCount;
    uint32_t* indices;
    uint32_t indicesCount;
    Weight* weights;
    uint32_t weightsCount;
};

#ifdef __HIPCC__
    struct __align__(16) Model3D
#else
    struct alignas(16) Model3D
#endif
{
    Joint* joints;
    uint32_t numJoints;
    ModelSubset* subsets;
    uint32_t numSubsets;
    ModelAnimation* animations;
    uint32_t numAnimations;
    float animationSpeedMultiplier;
    float vertexMagnitudeMultiplier;
    bool isSwapYZNeeded;
    uint32_t vertBytes;   // Size of the vertex type in bytes
    };
}  // namespace md5_cuda_animation

#ifndef defined(MD5_CUDA_VERBOSE_LOG)
    constexpr bool gpu_debug_enabled = true;
#else
    constexpr bool gpu_debug_enabled = false;    
#endif

__device__  __constant__ bool GPU_DEBUG_ENABLED;

//  Error checking macro
#define cudaCheckError(ans)                   \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define gpuKernelCheck()                     \
    {                                        \
        gpuKernelAssert(__FILE__, __LINE__); \
    }
inline void gpuKernelAssert(const char* file, int line, bool abort = true) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s %s %d\n", hipGetErrorString(err), file, line);
        if (abort)
            exit(err);
    }
}

namespace cuda {
// Find the GPU which is selected by Vulkan and supports CUDA
int getCudaDevice(uint8_t* vkDeviceUUID, size_t UUID_SIZE) {
    constexpr int INVALID_CUDA_DEVICE = -1;
    int current_device = 0;
    int device_count = 0;
    int devices_prohibited = 0;

    hipDeviceProp_t deviceProp;
    cudaCheckError(hipGetDeviceCount(&device_count));

    if (device_count == 0) {
        fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
        return INVALID_CUDA_DEVICE;
    }

    // Find the GPU which is selected by Vulkan
    while (current_device < device_count) {
        hipGetDeviceProperties(&deviceProp, current_device);

        if ((deviceProp.computeMode != hipComputeModeProhibited)) {
            // Compare the cuda device UUID with vulkan UUID
            int ret = memcmp((void*)&deviceProp.uuid, vkDeviceUUID, UUID_SIZE);
            if (ret == 0) {
                cudaCheckError(hipSetDevice(current_device));
                cudaCheckError(hipGetDeviceProperties(&deviceProp, current_device));
                printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", current_device, deviceProp.name,
                       deviceProp.major, deviceProp.minor);

                return current_device;
            }
        } else {
            devices_prohibited++;
        }

        current_device++;
    }

    if (devices_prohibited == device_count) {
        fprintf(stderr,
                "CUDA error:"
                " No Vulkan-CUDA Interop capable GPU found.\n");
        return INVALID_CUDA_DEVICE;
    }

    return INVALID_CUDA_DEVICE;
}
}  // namespace cuda

MD5CudaAnimation::MD5CudaAnimation(int cudaDevice, void* winMemHandleOfVkBufMem, uint64_t vkBufSize,
                                   void* winVkSemaphoreHandle, md5_animation::Model3D& _MD5Model, 
                                   uint64_t instancesBufferOffset, const std::vector<Instance>& instances, 
                                   float radius, bool isSwapYZNeeded, float animationSpeedMultiplier,
                                   float vertexMagnitudeMultiplier, uint64_t cuda_signalVkValue)
    : cpu_MD5Model(_MD5Model),
      cuda_signalVkValue(cuda_signalVkValue),
      cuda_instancesBufferOffset(instancesBufferOffset),
      cuda_radius(radius) {
    assert(_MD5Model.animations.size() > 0u && _MD5Model.subsets.size() > 0u &&
           _MD5Model.joints.size() > 0u);

    cudaCheckError(hipMalloc((void**)&cuda_ViewProj, sizeof(glm::mat4)));

    // import the Vulkan buffer memory to CUDA space
    {
        hipExternalMemory_t m_cudaExternalVKmem;

        hipExternalMemoryHandleDesc externalMemoryHandleDesc = {};
        externalMemoryHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
        externalMemoryHandleDesc.size = vkBufSize;  // Size of the external memory object
        externalMemoryHandleDesc.handle.win32.handle =
            winMemHandleOfVkBufMem;  // external win32 memory handle of m_generalBufferMemory

        cudaCheckError(hipImportExternalMemory(&m_cudaExternalVKmem, &externalMemoryHandleDesc));

        hipExternalMemoryBufferDesc externalMemBufferDesc = {};
        externalMemBufferDesc.offset = 0;
        externalMemBufferDesc.size = vkBufSize;  // Size of the external memory buffer
        externalMemBufferDesc.flags = 0;

        cudaCheckError(hipExternalMemoryGetMappedBuffer((void**)&cuda_extrVkMappedBuffer, m_cudaExternalVKmem, &externalMemBufferDesc));
    }

    // import the Vulkan semaphore to CUDA space
    hipExternalSemaphore_t cudaSem;
    {
        hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc = {};
        externalSemaphoreHandleDesc.type = cudaExternalSemaphoreHandleTypeTimelineSemaphoreWin32;
        externalSemaphoreHandleDesc.handle.win32.handle = winVkSemaphoreHandle;
        externalSemaphoreHandleDesc.flags = 0;

        cudaCheckError(hipImportExternalSemaphore(&cudaSem, &externalSemaphoreHandleDesc));
        cuda_semaphoreHandle = cudaSem;
    }

    cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(GPU_DEBUG_ENABLED), &gpu_debug_enabled, sizeof(gpu_debug_enabled)));

    {
        hipDeviceProp_t prop = {};
        cudaCheckError(hipSetDevice(cudaDevice));
        cudaCheckError(hipGetDeviceProperties(&prop, cudaDevice));

        cuda_warpSize = prop.warpSize;
        hipDeviceGetAttribute(&cuda_SMs, hipDeviceAttributeMultiprocessorCount, cudaDevice);

        cudaCheckError(hipStreamCreateWithFlags((hipStream_t*)&hip_stream, hipStreamNonBlocking));
    }

    // Allocate device memory for the model data
    cudaCheckError(hipMalloc((void**)&cuda_MD5Model, sizeof(md5_cuda_animation::Model3D)));

    for (size_t i = 0u; i < _MD5Model.animations.size(); ++i) {
        if (_MD5Model.animations[i].numJoints > cuda_maxJointsPerSkeleton) {
            cuda_maxJointsPerSkeleton = _MD5Model.animations[i].numJoints;
        }
    }
    // Allocate device memory for the interpolated skeleton
    cudaCheckError(hipMalloc((void**)&cuda_interpolatedSkeleton, cuda_maxJointsPerSkeleton * sizeof(md5_cuda_animation::Joint)));

    cuda_cleanupFunctions.push_back([this]() {
        cudaCheckError(hipFree(cuda_interpolatedSkeleton));
        cudaCheckError(hipFree(cuda_MD5Model));
        cudaCheckError(hipStreamDestroy((hipStream_t)hip_stream));
        cudaCheckError(hipDestroyExternalSemaphore((hipExternalSemaphore_t)cuda_semaphoreHandle));
        cudaCheckError(hipFree(cuda_extrVkMappedBuffer));
        cudaCheckError(hipFree(cuda_ViewProj));
    });

    md5_cuda_animation::Model3D host_MD5Model;

    host_MD5Model.numSubsets = static_cast<int>(_MD5Model.subsets.size());
    host_MD5Model.numJoints = static_cast<int>(_MD5Model.joints.size());
    host_MD5Model.numAnimations = static_cast<int>(_MD5Model.animations.size());
    host_MD5Model.joints = nullptr;
    host_MD5Model.subsets = nullptr;
    host_MD5Model.animations = nullptr;
    host_MD5Model.animationSpeedMultiplier = animationSpeedMultiplier;
    host_MD5Model.vertexMagnitudeMultiplier = vertexMagnitudeMultiplier;
    host_MD5Model.isSwapYZNeeded = isSwapYZNeeded;
    host_MD5Model.vertBytes = sizeof(VertexData);

    thrust::host_vector<md5_cuda_animation::Joint> joints(_MD5Model.joints.size());
    for (size_t i = 0u; i < _MD5Model.joints.size(); ++i) {
        joints[i] = md5_cuda_animation::Joint{_MD5Model.joints[i].parentID, _MD5Model.joints[i].pos, _MD5Model.joints[i].orientation};
    }

    // Allocate device memory for joints
    {
        size_t joints_size = joints.size() * sizeof(md5_cuda_animation::Joint);
        md5_cuda_animation::Joint* joints_device;

        cudaCheckError(hipMalloc((void**)&joints_device, joints_size));
        cudaCheckError(hipMemcpy(joints_device, joints.data(), joints_size, hipMemcpyHostToDevice));
        host_MD5Model.joints = joints_device;  // Set the pointer to the device memory

        cuda_cleanupFunctions.push_back([joints_device]() {
            cudaCheckError(hipFree(joints_device)); });
    }

    // Allocate device memory for subsets
    md5_cuda_animation::ModelSubset* subsets_device;
    size_t subset_size = sizeof(md5_cuda_animation::ModelSubset);
    size_t subsets_size = _MD5Model.subsets.size() * subset_size;
    hipMalloc((void**)&subsets_device, subsets_size);

    cuda_cleanupFunctions.push_back([subsets_device]() {
        cudaCheckError(hipFree(subsets_device));
    });

    thrust::host_vector<md5_cuda_animation::ModelSubset> subsets(_MD5Model.subsets.size());

    for (size_t i = 0u; i < _MD5Model.subsets.size(); ++i) {
        subsets[i] = md5_cuda_animation::ModelSubset{
            _MD5Model.subsets[i].numTriangles,
            _MD5Model.subsets[i].realMaterialId,
            _MD5Model.subsets[i].indexOffset,
            _MD5Model.subsets[i].vertOffset,
            nullptr,  // gpuVertices will be allocated later
            static_cast<uint32_t>(_MD5Model.subsets[i].gpuVertices.size()),
            nullptr,  // vertices will be allocated later
            static_cast<uint32_t>(_MD5Model.subsets[i].vertices.size()),
            nullptr,  // indices will be allocated later
            static_cast<uint32_t>(_MD5Model.subsets[i].indices.size()),
            nullptr,  // weights will be allocated later
            static_cast<uint32_t>(_MD5Model.subsets[i].weights.size())};

        // Copy vertices
        thrust::host_vector<md5_cuda_animation::MD5Vertex> vertices(_MD5Model.subsets[i].vertices.size());
        for (size_t j = 0u; j < _MD5Model.subsets[i].vertices.size(); ++j) {
            vertices[j] = md5_cuda_animation::MD5Vertex{_MD5Model.subsets[i].vertices[j].gpuVertexIndex, _MD5Model.subsets[i].vertices[j].startWeight,
                _MD5Model.subsets[i].vertices[j].weightCount};
        }

        // Copy vertices to device
        md5_cuda_animation::MD5Vertex* md5vertices_device;
        size_t md5vertices_size = vertices.size() * sizeof(md5_cuda_animation::MD5Vertex);
        cudaCheckError(hipMalloc((void**)&md5vertices_device, md5vertices_size));
        cudaCheckError(hipMemcpy(md5vertices_device, vertices.data(), md5vertices_size, hipMemcpyHostToDevice));
        subsets[i].vertices = md5vertices_device; // holds real device address

        thrust::host_vector<VertexData> gpuVertices(_MD5Model.subsets[i].gpuVertices.size());
        gpuVertices.assign(_MD5Model.subsets[i].gpuVertices.begin(), _MD5Model.subsets[i].gpuVertices.end());

        // Copy gpuVertices to device
        VertexData* gpuvertices_device;
        size_t gpuvertices_size = gpuVertices.size() * sizeof(VertexData);
        cudaCheckError(hipMalloc((void**)&gpuvertices_device, gpuvertices_size));
        cudaCheckError(hipMemcpy(gpuvertices_device, gpuVertices.data(), gpuvertices_size, hipMemcpyHostToDevice));
        subsets[i].gpuVertices = gpuvertices_device;  // holds real device address

        thrust::host_vector<uint32_t> indices(_MD5Model.subsets[i].indices.size());
        indices.assign(_MD5Model.subsets[i].indices.begin(), _MD5Model.subsets[i].indices.end());

        // Copy indices to device
        uint32_t* indices_device;
        size_t indices_size = indices.size() * sizeof(uint32_t);
        cudaCheckError(hipMalloc((void**)&indices_device, indices_size));
        cudaCheckError(hipMemcpy(indices_device, indices.data(), indices_size, hipMemcpyHostToDevice));
        subsets[i].indices = indices_device;  // holds real device address

        thrust::host_vector<md5_cuda_animation::Weight> weights(_MD5Model.subsets[i].weights.size());
        for (size_t j = 0u; j < _MD5Model.subsets[i].weights.size(); ++j) {
            weights[j] = md5_cuda_animation::Weight{
                _MD5Model.subsets[i].weights[j].jointID, _MD5Model.subsets[i].weights[j].bias, 
                _MD5Model.subsets[i].weights[j].pos, _MD5Model.subsets[i].weights[j].normal
            };
        }

        // Copy weights to device
        md5_cuda_animation::Weight* weights_device;
        size_t weights_size = weights.size() * sizeof(md5_cuda_animation::Weight);
        cudaCheckError(hipMalloc((void**)&weights_device, weights_size));
        cudaCheckError(hipMemcpy(weights_device, weights.data(), weights_size, hipMemcpyHostToDevice));
        subsets[i].weights = weights_device;  // holds real device address

        // Copy subsets[i] to device
        cudaCheckError(hipMemcpy(&subsets_device[i], &subsets[i], subset_size, hipMemcpyHostToDevice));
        
        cuda_cleanupFunctions.push_back([weights_device, indices_device, gpuvertices_device, md5vertices_device]() {
            cudaCheckError(hipFree(weights_device)); 
            cudaCheckError(hipFree(indices_device));
            cudaCheckError(hipFree(gpuvertices_device));
            cudaCheckError(hipFree(md5vertices_device));
        });
    }

    // Set the pointer to the device memory
    host_MD5Model.subsets = subsets_device;

    // Allocate device memory for animations
    md5_cuda_animation::ModelAnimation* animations_device;
    size_t animation_size = sizeof(md5_cuda_animation::ModelAnimation);
    size_t animations_size = _MD5Model.animations.size() * animation_size;
    hipMalloc((void**)&animations_device, animations_size);

    cuda_cleanupFunctions.push_back([animations_device]() {
        cudaCheckError(hipFree(animations_device));
    });

    thrust::host_vector<md5_cuda_animation::ModelAnimation> animations(_MD5Model.animations.size());

    for (size_t i = 0u; i < _MD5Model.animations.size(); ++i) {
        animations[i] = md5_cuda_animation::ModelAnimation{
            _MD5Model.animations[i].numFrames,
            _MD5Model.animations[i].numJoints,
            _MD5Model.animations[i].frameRate,
            _MD5Model.animations[i].numAnimatedComponents,
            _MD5Model.animations[i].frameTime,
            _MD5Model.animations[i].totalAnimTime,
            _MD5Model.animations[i].currAnimTime,
            nullptr,  // jointInfo will be allocated later
            static_cast<uint32_t>(_MD5Model.animations[i].jointInfo.size()),
            nullptr,  // frameBounds will be allocated later
            static_cast<uint32_t>(_MD5Model.animations[i].frameBounds.size()),
            nullptr,  // baseFrameJoints will be allocated later
            static_cast<uint32_t>(_MD5Model.animations[i].baseFrameJoints.size()),
            nullptr,  // frameData will be allocated later
            static_cast<uint32_t>(_MD5Model.animations[i].frameData.size()),
            nullptr,  // frameSkeleton will be allocated later
            static_cast<uint32_t>(_MD5Model.animations[i].frameSkeleton.size())};

        thrust::host_vector<md5_cuda_animation::AnimJointInfo> jointInfos(_MD5Model.animations[i].jointInfo.size());
        for (size_t j = 0u; j < _MD5Model.animations[i].jointInfo.size(); ++j) {
            jointInfos[j] = md5_cuda_animation::AnimJointInfo{_MD5Model.animations[i].jointInfo[j].parentID, _MD5Model.animations[i].jointInfo[j].flags,
                _MD5Model.animations[i].jointInfo[j].startIndex};
        }

        // Copy jointInfos to device
        md5_cuda_animation::AnimJointInfo* jointInfos_device;
        size_t jointInfos_size = jointInfos.size() * sizeof(md5_cuda_animation::AnimJointInfo);
        cudaCheckError(hipMalloc((void**)&jointInfos_device, jointInfos_size));
        cudaCheckError(hipMemcpy(jointInfos_device, jointInfos.data(), jointInfos_size, hipMemcpyHostToDevice));
        animations[i].jointInfo = jointInfos_device;

        thrust::host_vector<md5_cuda_animation::BoundingBox> frameBounds(_MD5Model.animations[i].frameBounds.size());
        for (size_t j = 0u; j < _MD5Model.animations[i].frameBounds.size(); ++j) {
            frameBounds[j] = md5_cuda_animation::BoundingBox{_MD5Model.animations[i].frameBounds[j].min,
                                                             _MD5Model.animations[i].frameBounds[j].max};
        }
        
        // Copy frameBounds to device
        md5_cuda_animation::BoundingBox* frameBounds_device;
        size_t frameBounds_size = frameBounds.size() * sizeof(md5_cuda_animation::BoundingBox);
        cudaCheckError(hipMalloc((void**)&frameBounds_device, frameBounds_size));
        cudaCheckError(hipMemcpy(frameBounds_device, frameBounds.data(), frameBounds_size, hipMemcpyHostToDevice));
        animations[i].frameBounds = frameBounds_device;

        thrust::host_vector<md5_cuda_animation::Joint> baseFrameJoints(_MD5Model.animations[i].baseFrameJoints.size());
        for (size_t j = 0u; j < _MD5Model.animations[i].baseFrameJoints.size(); ++j) {
            baseFrameJoints[j] = md5_cuda_animation::Joint{_MD5Model.animations[i].baseFrameJoints[j].parentID,
                _MD5Model.animations[i].baseFrameJoints[j].pos, _MD5Model.animations[i].baseFrameJoints[j].orientation};
        }

        // Copy baseFrameJoints to device
        md5_cuda_animation::Joint* baseFrameJoints_device;
        size_t baseFrameJoints_size = baseFrameJoints.size() * sizeof(md5_cuda_animation::Joint);
        cudaCheckError(hipMalloc((void**)&baseFrameJoints_device, baseFrameJoints_size));
        cudaCheckError(hipMemcpy(baseFrameJoints_device, baseFrameJoints.data(), baseFrameJoints_size, hipMemcpyHostToDevice));
        animations[i].baseFrameJoints = baseFrameJoints_device;

        thrust::host_vector<md5_cuda_animation::FrameData> frameData(_MD5Model.animations[i].frameData.size());
        for (size_t j = 0u; j < _MD5Model.animations[i].frameData.size(); ++j) {
            frameData[j] = md5_cuda_animation::FrameData{_MD5Model.animations[i].frameData[j].frameID,
                                                         _MD5Model.animations[i].frameData[j].frameData.data(),
                                                         static_cast<uint32_t>(_MD5Model.animations[i].frameData[j].frameData.size())};
        }

        // Copy frameData to device
        md5_cuda_animation::FrameData* frameData_device;
        size_t frameData_size = frameData.size() * sizeof(md5_cuda_animation::FrameData);
        cudaCheckError(hipMalloc((void**)&frameData_device, frameData_size));
        cudaCheckError(hipMemcpy(frameData_device, frameData.data(), frameData_size, hipMemcpyHostToDevice));
        animations[i].frameData = frameData_device;

        // Copy frameSkeleton to device
        md5_cuda_animation::Joint** frameSkeleton_device;
        size_t frameSkeleton_size = _MD5Model.animations[i].frameSkeleton.size() * sizeof(md5_cuda_animation::Joint*);
        cudaCheckError(hipMalloc((void**)&frameSkeleton_device, frameSkeleton_size));
        //cudaCheckError(hipMemcpy(&(animations_device[i].frameSkeleton), frameSkeleton_device, sizeof(md5_cuda_animation::Joint**), hipMemcpyHostToDevice));

        cuda_cleanupFunctions.push_back([jointInfos_device, frameData_device, baseFrameJoints_device, frameBounds_device]() {
            cudaCheckError(hipFree(jointInfos_device));
            cudaCheckError(hipFree(frameData_device));
            cudaCheckError(hipFree(baseFrameJoints_device));
            cudaCheckError(hipFree(frameBounds_device));
        });

        thrust::host_vector<md5_cuda_animation::Joint*> frameSkeleton(_MD5Model.animations[i].frameSkeleton.size());
        for (size_t j = 0u; j < _MD5Model.animations[i].frameSkeleton.size(); ++j) {
            thrust::host_vector<md5_cuda_animation::Joint> frameSkeletonJoints(_MD5Model.animations[i].frameSkeleton[j].size());
            for (size_t k = 0u; k < _MD5Model.animations[i].frameSkeleton[j].size(); ++k) {
                frameSkeletonJoints[k] = md5_cuda_animation::Joint{_MD5Model.animations[i].frameSkeleton[j][k].parentID,
                                                                   _MD5Model.animations[i].frameSkeleton[j][k].pos,
                                                                   _MD5Model.animations[i].frameSkeleton[j][k].orientation};
            }

            // Copy frameSkeletonJoints to device
            md5_cuda_animation::Joint* frameSkeletonJoints_device;
            size_t frameSkeletonJoints_size = frameSkeletonJoints.size() * sizeof(md5_cuda_animation::Joint);
            cudaCheckError(hipMalloc((void**)&frameSkeletonJoints_device, frameSkeletonJoints_size));
            cudaCheckError(hipMemcpy(frameSkeletonJoints_device, frameSkeletonJoints.data(), frameSkeletonJoints_size, hipMemcpyHostToDevice));
            frameSkeleton[j] = frameSkeletonJoints_device;

            cuda_cleanupFunctions.push_back([frameSkeletonJoints_device]() {
                cudaCheckError(hipFree(frameSkeletonJoints_device));
            });
        }

        cudaCheckError(hipMemcpy(frameSkeleton_device, frameSkeleton.data(), frameSkeleton_size, hipMemcpyHostToDevice));
        animations[i].frameSkeleton = frameSkeleton_device;

        // Copy animations[i] to device
        cudaCheckError(hipMemcpy(&animations_device[i], &animations[i], animation_size, hipMemcpyHostToDevice));
    }
    // Set the pointer to the device memory
    host_MD5Model.animations = animations_device;

    // Copying the host model to device except the pointers
    cudaCheckError(hipMemcpy(cuda_MD5Model, &host_MD5Model, sizeof(md5_cuda_animation::Model3D), hipMemcpyHostToDevice));


    // Update the device model with the initial data
    // Indices data is copied only once
    for (int32_t i = 0; i < cpu_MD5Model.numSubsets; i++) {
        auto& subset = cpu_MD5Model.subsets[i];

        hipDeviceSynchronize();  // Wait for kernel to be idle
        
        // Update the subset's buffer
        const uint32_t indexBytes = sizeof(subset.indices[0]);
        
        const uint32_t indicesSize = indexBytes * subset.indices.size();
        
        cudaCheckError(hipMemcpy((char*)cuda_extrVkMappedBuffer + subset.indexOffset * indexBytes, subset.indices.data(), indicesSize,
                   hipMemcpyHostToDevice));
    }

    // Instances data initialization
    {
        hipDeviceSynchronize();  // Wait for kernel to be idle
        const uint64_t instancesSize = sizeof(instances[0]) * instances.size();
        cudaCheckError(hipMalloc((void**)&cuda_instances_original, instancesSize));
        cudaCheckError(hipMalloc((void**)&cuda_instances_filtered, instancesSize));
        cudaCheckError(hipMemcpy(cuda_instances_original, instances.data(), instancesSize, hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy((char*)cuda_extrVkMappedBuffer + instancesBufferOffset, cuda_instances_original, instancesSize,
                                  hipMemcpyDeviceToDevice));
        cudaCheckError(hipMemcpy(cuda_instances_filtered, cuda_instances_original, instancesSize, hipMemcpyDeviceToDevice));
        cudaCheckError(hipMalloc((void**)&cuda_instances_flags, instancesSize));
        cudaCheckError(hipMemset(cuda_instances_flags, 0, instancesSize));

        cudaCheckError(hipHostMalloc((void**)&cuda_activeInstancesCount, sizeof(uint32_t)));   

        cuda_cleanupFunctions.push_back([this]() {
            cudaCheckError(hipFree(cuda_instances_original));
            cudaCheckError(hipFree(cuda_instances_filtered));
            cudaCheckError(hipFree(cuda_instances_flags));
            cudaCheckError(hipFree(cuda_activeInstancesCount));
        });

        cuda_numInstances = instances.size();
    }
}

__device__ void swapYandZ(glm::vec3& vertexData) {
    float temp = vertexData.y;
    vertexData.y = vertexData.z;
    vertexData.z = temp;
    vertexData.z *= -1.0f;
}

__global__ void updateAnimationChunk(md5_cuda_animation::Model3D* cuda_MD5Model,
                                     md5_cuda_animation::Joint* cuda_interpolatedSkeleton, int subsetId,
                                     char* cuda_extrVkMappedBuffer, uint64_t verticesBufferOffset) {
    // Unique thread index among all blocks
    int globalThreadIndx = threadIdx.x + blockDim.x * blockIdx.x;
    // thread index within one block
    int blockThreadXIndx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    /// if (GPU_DEBUG_ENABLED && globalThreadIndx == 0) {
    ///     printf("updateAnimationChunk subsetId:%d\n", subsetId);
    /// }
    if (cuda_MD5Model->numSubsets <= subsetId) {
        printf("updateAnimationChunk: subsetId is out of range\n");
        return;
    }
    md5_cuda_animation::ModelSubset& subset = cuda_MD5Model->subsets[subsetId];

    /** Note use can use cuda_MD5Model.indexBytes and cuda_MD5Model.vertBytes instead of the shared memory to calculate and sync ecery time
    __shared__ uint32_t indexBytes;
    __shared__ uint32_t vertBytes;

    // init shared data on the first thread for each SM block
    if (blockThreadXIndx == 0 && cuda_MD5Model.indexBytes == 0) {
        indexBytes = sizeof(subset.indices[0]);
        vertBytes = sizeof(subset.gpuVertices[0]);
    }

    // Synchronize threads within the warp to ensure all threads have the same indexBytes, vertBytes values
    __syncwarp();*/

    // Note: we have more indices than vertices, so we need to skip the globalThreadIndx that are out of bounds

    // Update the subset's buffer by copying i-th vertex data to the mapped buffer
    if (subset.gpuVerticesCount > globalThreadIndx) {
        glm::vec3 rotatedPoint = glm::vec3(.0f, .0f, .0f);
        md5_cuda_animation::MD5Vertex& tempVert = subset.vertices[globalThreadIndx];
        VertexData& gpuVertex = subset.gpuVertices[tempVert.gpuVertexIndex];
        gpuVertex.pos = glm::vec3(.0f, .0f, .0f);     // Make sure the vertex's pos is cleared first
        gpuVertex.normal = glm::vec3(.0f, .0f, .0f);  // Clear vertices normal

        // Sum up the joints and weights information to get vertex's position and normal
        for (uint32_t j = 0; j < tempVert.weightCount; ++j) {
            const md5_cuda_animation::Weight& tempWeight = subset.weights[tempVert.startWeight + j];
            const md5_cuda_animation::Joint& tempJoint = cuda_interpolatedSkeleton[tempWeight.jointID];

            // Calculate vertex position (in joint space, eg. rotate the point around (0,0,0)) for this weight using the joint
            // orientation quaternion and its conjugate We can rotate a point using a quaternion with the equation
            // "rotatedPoint = quaternion * point * quaternionConjugate" but conjugate id nor actual for glm since it has
            // internal optimization
            rotatedPoint = tempJoint.orientation * tempWeight.pos;

            // Now move the verices position from joint space (0,0,0) to the joints position in world space, taking the
            // weights bias into account
            gpuVertex.pos += (tempJoint.pos + rotatedPoint) * tempWeight.bias;

            // Compute the normals for this frames skeleton using the weight normals from before
            // We can comput the normals the same way we compute the vertices position, only we don't have to translate them
            // (just rotate)
            rotatedPoint = tempJoint.orientation * tempWeight.normal;

            // Add to vertices normal and take weight bias into account
            gpuVertex.normal = gpuVertex.normal + (rotatedPoint * tempWeight.bias);
        }

        gpuVertex.pos *= cuda_MD5Model->vertexMagnitudeMultiplier;

        gpuVertex.normal = glm::normalize(gpuVertex.normal);

        if (cuda_MD5Model->isSwapYZNeeded) {
            swapYandZ(gpuVertex.pos);
            swapYandZ(gpuVertex.normal);
        }

        memcpy(cuda_extrVkMappedBuffer + verticesBufferOffset + (subset.vertOffset + globalThreadIndx) * cuda_MD5Model->vertBytes,
               &subset.gpuVertices[globalThreadIndx], cuda_MD5Model->vertBytes);
    } 
    
    // Note: we don't need to update indices every time, since they are static and already copied to the mapped buffer
    // Update the subset's buffer by copying i-th index to the mapped buffer
    /*if (globalThreadIndx < subset.indicesCount) {
        memcpy(cuda_extrVkMappedBuffer + (subset.indexOffset + globalThreadIndx) * cuda_MD5Model->indexBytes,
               &subset.indices[globalThreadIndx], cuda_MD5Model->indexBytes);
    }*/
}

__device__ void calculateInterpolatedSkeleton(md5_cuda_animation::Model3D* cuda_MD5Model, int animationID,
                                              md5_cuda_animation::Joint* cuda_interpolatedSkeleton, int frame0, int frame1,
                                              float interpolation) {
    // Unique thread index among all blocks
    int globalThreadIndx = threadIdx.x + blockDim.x * blockIdx.x;
    ///if (GPU_DEBUG_ENABLED && globalThreadIndx == 0) {
    ///    printf("calculateInterpolatedSkeleton animationID:%d; frame0: %d; frame1: %d; interpolation: %f\n", animationID, frame0,
    ///           frame1, interpolation);
    ///}

    md5_cuda_animation::ModelAnimation& animation = cuda_MD5Model->animations[animationID];
    ///if (GPU_DEBUG_ENABLED && globalThreadIndx == 0) {
    ///    printf("Current Model3D: numJoints: %d; numSubsets: %d; numAnimations: %d\n", cuda_MD5Model->numJoints,
    ///           cuda_MD5Model->numSubsets, cuda_MD5Model->numAnimations);
    ///}
    if (0 > animation.numJoints || animation.frameSkeletonCount <= frame0 || animation.frameSkeletonCount <= frame1) {
        printf("out of range\n");
        return;
    }

    md5_cuda_animation::Joint joint0;
    md5_cuda_animation::Joint joint1;
    if (globalThreadIndx < animation.numJoints) {
        md5_cuda_animation::Joint& tempJoint = cuda_interpolatedSkeleton[globalThreadIndx];  // Use globalThreadIndx for indexing
        joint0 = animation.frameSkeleton[frame0][globalThreadIndx];
        joint1 = animation.frameSkeleton[frame1][globalThreadIndx];

        tempJoint.parentID = joint0.parentID;  // Set the tempJoints parent id

        // Interpolate positions
        tempJoint.pos = joint0.pos + (interpolation * (joint1.pos - joint0.pos));

        // Interpolate orientations using spherical interpolation (Slerp)
        tempJoint.orientation = glm::slerp(joint0.orientation, joint1.orientation, interpolation);

        // joint updating of our interpolated skeleton completed
    }
}

__global__ void cuda_md5_update(md5_cuda_animation::Model3D* cuda_MD5Model,
                                md5_cuda_animation::Joint* cuda_interpolatedSkeleton, float deltaTimeMS, int animationID) {
    // Unique thread index among all blocks
    int globalThreadIndx = threadIdx.x + blockDim.x * blockIdx.x;
    // thread index within one block
    int blockThreadXIndx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    if (cuda_MD5Model->numAnimations <= animationID) {
        if (globalThreadIndx == 0)
            printf("wrong animationID: %d\n", animationID);
        return;
    }

    if (cuda_MD5Model->animations[animationID].numFrames <= 1) {
        if (globalThreadIndx == 0)
            printf("numFrames <= 1\n");
        return;
    }

    ///if (GPU_DEBUG_ENABLED && globalThreadIndx == 0) {
    ///    printf("Updating animation %d with deltaTimeMS: %f\n", animationID, deltaTimeMS);
    ///    printf("currAnimTime: %f\n", cuda_MD5Model->animations[animationID].currAnimTime);
    ///    printf("Current Model3D: numJoints: %d; numSubsets: %d; numAnimations: %d\n", 
    ///           cuda_MD5Model->numJoints, 
    ///           cuda_MD5Model->numSubsets,
    ///           cuda_MD5Model->numAnimations);
    ///}

    __shared__ float currentFrame;
    __shared__ int frame0;
    __shared__ int frame1;
    __shared__ float interpolation;
    __shared__ float currAnimTime;

    // init shared data on the first thread for each SM block
    if (blockThreadXIndx == 0) {
        currAnimTime = cuda_MD5Model->animations[animationID].currAnimTime +
                       cuda_MD5Model->animationSpeedMultiplier * deltaTimeMS / 1000.0f;  // Update the current animation time

        if (currAnimTime >= cuda_MD5Model->animations[animationID].totalAnimTime)
            currAnimTime = 0.0f;

        // Which frame are we on
        currentFrame = currAnimTime * cuda_MD5Model->animations[animationID].frameRate;
        frame0 = static_cast<int>(floorf(currentFrame));
        frame1 = frame0 + 1;

        // Make sure we don't go over the number of frames
        if (frame0 == cuda_MD5Model->animations[animationID].numFrames - 1)
            frame1 = 0;

        interpolation =
            currentFrame - frame0;  // Get the remainder (in time) between frame0 and frame1 to use as interpolation factor
    }

    // Synchronize threads within the warp to ensure all threads have the same currentFrame, frame0, frame1, and interpolation values
    __syncwarp();

    // each thread will calculate its own joint in the interpolatedSkeleton
    calculateInterpolatedSkeleton(cuda_MD5Model, animationID, cuda_interpolatedSkeleton, frame0, frame1, interpolation);

    // Synchronize threads to ensure all threads have completed the calculation of i'th joint before proceeding
    __syncthreads();

    // Update the current animation time in the model only once per animation update
    if (globalThreadIndx == 0) {
        cuda_MD5Model->animations[animationID].currAnimTime = currAnimTime;
    }

    // Print out the 10th joint of the interpolated skeleton for debugging purposes
    // check if the interpolatedSkeleton is synced for the next thread
    //if (GPU_DEBUG_ENABLED && globalThreadIndx == 1) {
    //    md5_cuda_animation::Joint& tempJoint = cuda_interpolatedSkeleton[10];
    //    printf("GPU(Cuda) InterpolatedSkeleton[10].parentID = %d; InterpolatedSkeleton[10].orientation = %f %f %f\n", tempJoint.parentID,
    //           tempJoint.orientation.x, tempJoint.orientation.y, tempJoint.orientation.z);
    //}
}

__global__ void cuda_filter_instances(uint32_t* out_activeInstancesCount, Instance* cuda_instances_original, uint32_t* cuda_instances_flags, 
                                      Instance* cuda_instances_filtered, glm::mat4* cuda_viewProj, uint32_t cuda_numInstances, char* cuda_extrVkMappedBuffer,
                                      uint64_t cuda_instancesBufferOffset, float z_far, float radius) {
    // Unique thread index among all blocks
    int globalThreadIndx = threadIdx.x + blockDim.x * blockIdx.x;
    // thread index within one block
    int blockThreadXIndx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    if (globalThreadIndx >= cuda_numInstances) {
        return;  // Out of bounds
    }

    glm::mat4& viewProj = *cuda_viewProj;

    __shared__ float biasValue;
    __shared__ glm::vec4 biasCubeValues[9];

    // init shared data on the first thread for each SM block
    if (blockThreadXIndx == 0) {
        biasValue = radius + 0.15f * z_far;  // to avoid choppy clipping of the model edges nearby the  camera
        biasCubeValues[0] = viewProj * glm::vec4(-biasValue, -biasValue, -biasValue, 1.0f);  // -Y
        biasCubeValues[1] = viewProj * glm::vec4(biasValue, -biasValue, -biasValue, 1.0f);
        biasCubeValues[2] = viewProj * glm::vec4(-biasValue, -biasValue, biasValue, 1.0f);
        biasCubeValues[3] = viewProj * glm::vec4(biasValue, -biasValue, biasValue, 1.0f);
        biasCubeValues[4] = viewProj * glm::vec4(-biasValue, biasValue, -biasValue, 1.0f);  // +Y
        biasCubeValues[5] = viewProj * glm::vec4(biasValue, biasValue, -biasValue, 1.0f);
        biasCubeValues[6] = viewProj * glm::vec4(-biasValue, biasValue, biasValue, 1.0f);
        biasCubeValues[7] = viewProj * glm::vec4(biasValue, biasValue, biasValue, 1.0f);
        biasCubeValues[8] = glm::vec4(0.0f, 0.0f, 0.0f, 0.0f);  // no bias (for center point)
    }

    // Synchronize threads within the warp to ensure all threads have the same bias values
    __syncwarp();

    const float maxLimitVal = 1.0f + FLT_EPSILON;  // float epsilon is used to avoid precision issues
    Instance& instance = cuda_instances_original[globalThreadIndx];
    glm::vec4 clipOrig = viewProj * glm::vec4(instance.posShift, 1.0f);
    cuda_instances_flags[globalThreadIndx] = 0;
    for (const auto& bias : biasCubeValues) {
        glm::vec4 clip = clipOrig + instance.scale * bias;
        glm::vec3 ndc = glm::vec3(clip.x / clip.w, clip.y / clip.w, clip.z / clip.w);
        // z is in range [0, 1] for NDC, so we can check it against 0.0f and maxLimitVal
        if (glm::abs(ndc.x) <= maxLimitVal && glm::abs(ndc.y) <= maxLimitVal && ndc.z <= maxLimitVal &&
            ndc.z >= 0.0f - FLT_EPSILON) {
            cuda_instances_flags[globalThreadIndx] = 1;  // visible instance
            /** Note: we don't have to use 'break' since gpu driver can not understand what thread of warp is stopped, it affects warp overall
            * break; 
            */
        }
    }
    // Synchronize threads to ensure all threads have completed the calculation of i'th instance before proceeding
    __syncthreads();

    // Update the filtered instances buffer
    if (globalThreadIndx == 0) {
        uint32_t visibleInstances = 0;
        for (uint32_t i = 0; i < cuda_numInstances; i++) {
            if (cuda_instances_flags[i] == 1) {
                cuda_instances_filtered[visibleInstances] = cuda_instances_original[i];
                visibleInstances++;
            }
        }
        const uint64_t instancesSize = sizeof(Instance) * visibleInstances;
        memcpy((char*)cuda_extrVkMappedBuffer + cuda_instancesBufferOffset, (char*)cuda_instances_filtered, instancesSize);
        *out_activeInstancesCount = visibleInstances;  // Update the count of active instances
    }
}

uint32_t MD5CudaAnimation::update(float deltaTimeMS, int animationID, uint64_t verticesBufferOffset, const glm::mat4& viewProj,
                              float z_far) {
    assert(cuda_MD5Model != nullptr && cuda_interpolatedSkeleton != nullptr && cuda_maxJointsPerSkeleton > 0u &&
           cpu_MD5Model.animations.size() > animationID);

    cudaCheckError(hipMemcpy(cuda_ViewProj, &viewProj[0][0], sizeof(glm::mat4), hipMemcpyHostToDevice));

    int threadsPerBlock = cuda_warpSize;
    int blocksPerGrid = cuda_SMs;

    // Filter instances based on the view projection matrix and z_far
    if (cuda_numInstances > 1u) {
        blocksPerGrid = cuda_numInstances / threadsPerBlock + 1;
        cuda_filter_instances<<<blocksPerGrid, threadsPerBlock, 0, (hipStream_t)hip_stream>>>(
            cuda_activeInstancesCount, cuda_instances_original, cuda_instances_flags, cuda_instances_filtered, cuda_ViewProj,
            cuda_numInstances, cuda_extrVkMappedBuffer, cuda_instancesBufferOffset, z_far, cuda_radius);
        gpuKernelCheck();
    }

    blocksPerGrid = cpu_MD5Model.animations[animationID].numJoints / threadsPerBlock + 1;
    cuda_md5_update<<<blocksPerGrid, threadsPerBlock, 0, (hipStream_t)hip_stream>>>(cuda_MD5Model, cuda_interpolatedSkeleton,
                                                                                      deltaTimeMS, animationID);
    gpuKernelCheck();
    hipDeviceSynchronize();  // Wait for cuda_interpolatedSkeleton completion before updating the subsets

    uint32_t activeInstancesCount = *cuda_activeInstancesCount;

    for (int32_t i = 0; i < cpu_MD5Model.numSubsets; i++) {
        auto& subset = cpu_MD5Model.subsets[i];
        int32_t subsetVertices = glm::max(subset.gpuVertices.size(),
                     subset.vertices.size());  // subset.indices.size() is not used since indices are copied only once
        blocksPerGrid = subsetVertices / threadsPerBlock + 1;
        updateAnimationChunk<<<blocksPerGrid, threadsPerBlock, 0, (hipStream_t)hip_stream>>>(
            cuda_MD5Model, cuda_interpolatedSkeleton, i, cuda_extrVkMappedBuffer, verticesBufferOffset);

        gpuKernelCheck();
    }

    // Signal vulkan to continue with the updated buffers
    hipExternalSemaphoreSignalParams signalParams = {};
    signalParams.flags = 0;
    signalParams.params.fence.value = cuda_signalVkValue;
    hipExternalSemaphore_t cudaSem = (hipExternalSemaphore_t)cuda_semaphoreHandle;
    cudaCheckError(hipSignalExternalSemaphoresAsync(&cudaSem, &signalParams, 1, (hipStream_t)hip_stream));
    cuda_signalVkValue++;  // increment signal value for next synchronization with next Vulkan swapchain

    return activeInstancesCount;
}
